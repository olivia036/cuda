#include "hip/hip_runtime.h"
#include "kernel.cuh"

//fitness_function define
__device__ float fitness_function(float x[]);

//setup random number
__global__ void setup_rand(hiprandState * state, unsigned long seed)
{
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	hiprand_init(seed, id, 0, &state[id]);
}


__global__ void kernelUpdateParticle(float *position, float *velocities, float *pBests, float *gBests, hiprandState* globalState1, hiprandState* globalState2)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;

	if (i >= NUM_OF_PARTICLES*NUM_OF_DIMENSIONS)
		return;

	hiprandState localState1 = globalState1[i];
	hiprandState localState2 = globalState2[i];
	//float rp = r1;
	//float rg = r2;
	float r1 = hiprand_uniform(&localState1);
	float r2 = hiprand_uniform(&localState2);
	globalState1[i] = localState1;
	globalState2[i] = localState2;
	//__syncthreads();
	velocities[i] = OMEGA*velocities[i] + c1*r1*(pBests[i] - position[i]) + c2*r2*(gBests[i] - position[i]);

	//Update position of particle
	position[i] += velocities[i];
}

__global__ void kernelUpdatePBest(float *positions, float *pBests, float *gBest)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;

	if (i >= NUM_OF_PARTICLES*NUM_OF_DIMENSIONS || i%NUM_OF_DIMENSIONS != 0)
		return;

	for (int j = 0; j < NUM_OF_DIMENSIONS; j++)
	{
		tempParticle1[i] = positions[i + j];
		tempParticle2[i] = pBests[i + j];
	}

	if (fitness_function(tempParticle1) < fitness_function(tempParticle2))
	{
		for (int k = 0; k < NUM_OF_DIMENSIONS; k++)
		{
			pBests[i + k] = positions[i + k];
		}
	}
}



//cuda_pso define
void cuda_pso(float *positions, float *velocities, float *pBests, float *gBest)
{
	int size = NUM_OF_PARTICLES*NUM_OF_DIMENSIONS;
	int memcpyCount = 1040;
	
	//setup a cuda stream  
	hipStream_t stream1;
	hipStreamCreate(&stream1);

	hipStream_t stream2;
	hipStreamCreate(&stream2);

	//setup random number seed
	hiprandState* devStates1;
	hiprandState* devStates2;
	
	hipMalloc(&devStates1, size * sizeof(hiprandState));
	hipMalloc(&devStates2, size * sizeof(hiprandState));

	//device memory allocation
	float *devPos1;
	float *devVel1;
	float *devPBest1;
	float *devGBest1;

	float *devPos2;
	float *devVel2;
	float *devPBest2;
	float *devGBest2;
	
	float temp[NUM_OF_DIMENSIONS];
	//stream1
	hipMalloc((void**)&devPos1, sizeof(float)*size/2);//GPU memory allocation
	hipMalloc((void**)&devVel1, sizeof(float)*size/2);
	hipMalloc((void**)&devPBest1, sizeof(float)*size/2);
	hipMalloc((void**)&devGBest1, sizeof(float)*size/2);
	//stream2
	hipMalloc((void**)&devPos2, sizeof(float)*size / 2);//GPU memory allocation
	hipMalloc((void**)&devVel2, sizeof(float)*size / 2);
	hipMalloc((void**)&devPBest2, sizeof(float)*size / 2);
	hipMalloc((void**)&devGBest2, sizeof(float)*size / 2);
	//CPU memory allocation
	hipHostAlloc((void**)&positions, sizeof(float)*size, hipHostMallocDefault);
	hipHostAlloc((void**)&velocities, sizeof(float)*size, hipHostMallocDefault);
	hipHostAlloc((void**)&pBests, sizeof(float)*size, hipHostMallocDefault);
	hipHostAlloc((void**)&gBest, sizeof(float)*size, hipHostMallocDefault);

	//set thread & block number
	int threadsNum = 256;
	int blocksNum = NUM_OF_PARTICLES / threadsNum;

	for (int i=0;i<size/2;i+=memcpyCount/2)
	{
		hipMemcpyAsync(devPos1, positions + i, sizeof(float)*size/2, hipMemcpyHostToDevice, stream1);
		hipMemcpyAsync(devVel1, positions + i, sizeof(float)*size/2, hipMemcpyHostToDevice, stream1);
		hipMemcpyAsync(devPBest1, positions + i, sizeof(float)*size/2, hipMemcpyHostToDevice, stream1);
		hipMemcpyAsync(devGBest1, positions + i, sizeof(float)*size/2, hipMemcpyHostToDevice, stream1);

		hipMemcpyAsync(devPos2, positions + i + memcpyCount, sizeof(float)*size / 2, hipMemcpyHostToDevice, stream2);
		hipMemcpyAsync(devVel2, positions + i + memcpyCount, sizeof(float)*size / 2, hipMemcpyHostToDevice, stream2);
		hipMemcpyAsync(devPBest2, positions + i + memcpyCount, sizeof(float)*size / 2, hipMemcpyHostToDevice, stream2);
		hipMemcpyAsync(devGBest2, positions + i + memcpyCount, sizeof(float)*size / 2, hipMemcpyHostToDevice, stream2);
	}
	hipStreamSynchronize(stream1);
	hipStreamSynchronize(stream2);

	//PSO main function
	for (int iter = 0; iter < MAX_ITER; iter++)
	{

		//initialize the random num
		setup_rand << < blocksNum, threadsNum >> > (devStates1, time(NULL));
		setup_rand << < blocksNum, threadsNum >> > (devStates2, time(NULL));

		//clock_t countBegin = clock();
		//Update position and velocity

		kernelUpdateParticle << <blocksNum, threadsNum >> > (devPos1, devVel1, devPBest1, devGBest1, devStates1, devStates2);
		kernelUpdateParticle << <blocksNum, threadsNum >> > (devPos2, devVel2, devPBest2, devGBest2, devStates2, devStates2);

		//Update pBest
		kernelUpdatePBest << <blocksNum, threadsNum >> > (devPos1, devPBest1, devGBest1);
		kernelUpdatePBest << <blocksNum, threadsNum >> > (devPos2, devPBest2, devGBest2);

		//Update gBest
		hipMemcpyAsync(pBests, devPBest1, sizeof(float)*NUM_OF_PARTICLES*NUM_OF_DIMENSIONS, hipMemcpyDeviceToHost,stream1);
		hipMemcpyAsync(pBests+memcpyCount, devPBest2, sizeof(float)*NUM_OF_PARTICLES*NUM_OF_DIMENSIONS, hipMemcpyDeviceToHost, stream1);

		for (int i = 0; i < size; i += NUM_OF_DIMENSIONS)
		{
			for (int k = 0; k < NUM_OF_DIMENSIONS; k++)
			{
				temp[k] = pBests[i + k];
			}

			if (host_fitness_function(temp) < host_fitness_function(gBest))
			{
				for (int k = 0; k < NUM_OF_DIMENSIONS; k++)
				{
					gBest[k] = temp[k];
				}
			}
		}

		hipMemcpyAsync(devGBest1, gBest, sizeof(float)*NUM_OF_DIMENSIONS, hipMemcpyDeviceToHost,stream2);
		hipMemcpyAsync(devGBest2, gBest, sizeof(float)*NUM_OF_DIMENSIONS, hipMemcpyDeviceToHost,stream2);

		//clock_t countEnd = clock();
		//printf("The iter time consumption : %10.3lf ms\n", (double)(countEnd - countBegin) / CLOCKS_PER_SEC);
		//printf("The iter number is: %d\n", iter);
	}

	for (int i = 0; i < size/2; i += memcpyCount/2)
	{
		hipMemcpyAsync(positions + i, devPos1 + i, sizeof(float)*size, hipMemcpyDeviceToHost, stream1);
		hipMemcpyAsync(positions + i, devVel1 + i, sizeof(float)*size, hipMemcpyDeviceToHost, stream1);
		hipMemcpyAsync(positions + i, devPBest1 + i, sizeof(float)*size, hipMemcpyDeviceToHost, stream1);
		hipMemcpyAsync(positions + i, devGBest1 + i, sizeof(float)*size, hipMemcpyDeviceToHost, stream1);

		hipMemcpyAsync(positions + i + memcpyCount, devPos1 + i + memcpyCount, sizeof(float)*size, hipMemcpyDeviceToHost, stream2);
		hipMemcpyAsync(positions + i + memcpyCount, devVel1 + i + memcpyCount, sizeof(float)*size, hipMemcpyDeviceToHost, stream2);
		hipMemcpyAsync(positions + i + memcpyCount, devPBest1 + i + memcpyCount, sizeof(float)*size, hipMemcpyDeviceToHost, stream2);
		hipMemcpyAsync(positions + i + memcpyCount, devGBest1 + i + memcpyCount, sizeof(float)*size, hipMemcpyDeviceToHost, stream2);

	}
	hipStreamSynchronize(stream1);
	hipStreamSynchronize(stream2);

	//clean up
	hipFree(devPos1);
	hipFree(devVel1);
	hipFree(devPBest1);
	hipFree(devGBest1);

	hipFree(devPos2);
	hipFree(devVel2);
	hipFree(devPBest2);
	hipFree(devGBest2);

	hipFree(devStates1);
	hipFree(devStates2);
}



//fitness_function define
__device__ float fitness_function(float x[])
{
	float res = 0;
	float y1 = 1 + (x[0] - 1) / 4;
	float yn = 1 + (x[NUM_OF_DIMENSIONS - 1] - 1) / 4;

	res += pow(sin(phi * y1), 2);

	for (int i = 0; i < NUM_OF_DIMENSIONS - 1; i++)
	{
		float y = 1 + (x[i] - 1) / 4;
		float yp = 1 + (x[i + 1] - 1) / 4;

		res += pow(y - 1, 2) * (1 + 10 * pow(sin(phi * yp), 2)) + pow(yn - 1, 2);

	}

	return res;
}

